#include <hip/hip_runtime.h>
#include <assert.h>
#include <cstdlib>
#include "common/matrix.h"
#include "common/eval.h"
#include "common/check.h"
#include <iostream>
#include <hipblas.h>

int main(int argc,char* argv[]) {
    int dim =1024;
    if(argc>1) {
        dim = atoi(argv[1]);
    }
    assert(dim%128==0);

    Matrix A,B,C;
    initializeMatrix(&A, dim);
    initializeMatrix(&B, dim);
    initializeMatrix(&C, dim);
    Eval eval;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float duration=eval.eval([&]() {
        float alpha = 1.0f;
        float beta = 0.0f;
        hipblasSgemm(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            dim, dim, dim,
            &alpha,
            B.elements, dim,
            A.elements, dim,
            &beta,
            C.elements, dim
        );
    });
    hipblasDestroy(handle);
    std::cout<<"duration: "<<duration<<" ms"<<std::endl;
    std::cout<<hipGetErrorString(hipGetLastError())<<std::endl;
    std::cout<<"isCalculationRight:"<<check(A,B,C)<<std::endl;

    freeMatrix(&A);
    freeMatrix(&B);
    freeMatrix(&C);
}