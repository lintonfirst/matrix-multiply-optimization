#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <cstdlib>
#include "common/matrix.h"
#include "common/eval.h"
#include "common/check.h"
#include <iostream>

#define TILE_SIZE 16

__global__ void matmul(Matrix A, Matrix B, Matrix C) {
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    float* matA = A.elements;
    float* matB = B.elements;
    float* matC = C.elements;

    int tile_id_row = blockIdx.y;
    int tile_id_col = blockIdx.x;
    int tile_offset_row = threadIdx.y;
    int tile_offset_col = threadIdx.x;
    float sum = 0.0f;
    for(int i=0;i<gridDim.x;i++){
        tile_A[tile_offset_row][tile_offset_col] = matA[(i*TILE_SIZE+tile_offset_col)+A.width*(tile_id_row*TILE_SIZE+tile_offset_row)];
        tile_B[tile_offset_row][tile_offset_col] = matB[(tile_id_col*TILE_SIZE+tile_offset_col)+B.width*(i*TILE_SIZE+tile_offset_row)];
        __syncthreads();
        for(int j=0;j<TILE_SIZE;j++) {
            sum += tile_A[tile_offset_row][j] * tile_B[j][tile_offset_col];
        }
        __syncthreads();
    }
    matC[(tile_id_row*TILE_SIZE+tile_offset_row)*C.width + (tile_id_col*TILE_SIZE+tile_offset_col)] = sum;
}



void basic_tiled_matmul(int dim,Matrix A,Matrix B,Matrix C){
    Eval eval;

    float duration=eval.eval([&]() {
        hipMemset(C.elements,0,dim*dim*sizeof(float));
        dim3 grid(dim/TILE_SIZE, dim/TILE_SIZE);
        dim3 block(TILE_SIZE,TILE_SIZE);
        matmul<<<grid,block>>>(A,B,C);
    });
    std::cout<<"Section: basic_tiled_matmul================================================="<<std::endl;
    std::cout<<"duration: "<<duration<<" ms"<<std::endl;
    std::cout<<hipGetErrorString(hipGetLastError())<<std::endl;
    std::cout<<"isCalculationRight:"<<check(A,B,C)<<std::endl;
}

__global__ void matmul2(Matrix A, Matrix B, Matrix C) {
    __shared__ float tile_A[64][64];
    __shared__ float tile_B[64][64];


    float* matA = A.elements;
    float* matB = B.elements;
    float* matC = C.elements;

    int tile_id_row = blockIdx.y*64;
    int tile_id_col = blockIdx.x*64;
    int tile_offset_row = threadIdx.y*4;
    int tile_offset_col = threadIdx.x*4;
    float sum[16];

    #pragma unroll
    for(int i=0;i<16;i++){
        sum[i] = 0;
    }

    for(int i=0;i<gridDim.x;i++){
        for(int j=0;j<4;j++){
            int smem_row = tile_offset_row + j;
            int smem_col = tile_offset_col;

            int global_row_A = tile_id_row + smem_row;
            int global_col_A = i * 64 + smem_col;

            float4* vec_tileA = (float4*)&tile_A[smem_row][smem_col];
            float4* vecA = (float4*)&matA[global_col_A + global_row_A * A.width];
            *vec_tileA = *vecA;

            int global_row_B = i * 64 + smem_row;
            int global_col_B = tile_id_col + smem_col;

            float4* vec_tileB = (float4*)&tile_B[smem_row][smem_col];
            float4* vecB = (float4*)&matB[global_col_B + global_row_B * B.width];
            *vec_tileB = *vecB;
        }
        __syncthreads();
        for(int j=0;j<4;j++) {
            int a_row = tile_offset_row  + j;
            for (int t = 0; t < 64; t++) {
                float a = tile_A[a_row][t];
                float4 vecB = *(float4*)&tile_B[t][tile_offset_col];

                sum[j * 4 + 0] += a * vecB.x;
                sum[j * 4 + 1] += a * vecB.y;
                sum[j * 4 + 2] += a * vecB.z;
                sum[j * 4 + 3] += a * vecB.w;
            }
        }
        __syncthreads();
    }
    for(int i=0;i<4;i++){
        int row = tile_id_row + tile_offset_row + i;
        int col = tile_id_col + tile_offset_col;
        float4* vecC = (float4*)&matC[row * C.width + col];
        *vecC = *(float4*)&sum[i * 4];
    }
}

void fragent_vec_tiled_matmul(int dim,Matrix A,Matrix B,Matrix C){
    Eval eval;
    float duration=eval.eval([&]() {
        dim3 grid(dim/64, dim/64);
        dim3 block(16,16);
        matmul2<<<grid,block>>>(A,B,C);
    });
    std::cout<<"Section:fragemnt_tiled_matmul================================================="<<std::endl;
    std::cout<<"duration: "<<duration<<" ms"<<std::endl;
    std::cout<<hipGetErrorString(hipGetLastError())<<std::endl;
    std::cout<<"isCalculationRight:"<<check(A,B,C)<<std::endl;
}

__global__ void matmul3(Matrix A, Matrix B, Matrix C) {
    __shared__ float tile_A[64][64];
    __shared__ float tile_B[64][64];


    float* matA = A.elements;
    float* matB = B.elements;
    float* matC = C.elements;

    int tile_id_row = blockIdx.y*64;
    int tile_id_col = blockIdx.x*64;
    int tile_offset_row = threadIdx.y*4;
    int tile_offset_col = threadIdx.x*4;
    float sum[16];

    #pragma unroll
    for(int i=0;i<16;i++){
        sum[i] = 0;
    }

    for(int i=0;i<gridDim.x;i++){
        for(int j=0;j<4;j++){
            int smem_row = tile_offset_row + j;
            int smem_col = tile_offset_col;

            int global_row_A = tile_id_row + smem_row;
            int global_col_A = i * 64 + smem_col;

            float4* vec_tileA = (float4*)&tile_A[smem_row][smem_col];
            float4* vecA = (float4*)&matA[global_col_A + global_row_A * A.width];
            *vec_tileA = *vecA;

            int global_row_B = i * 64 + smem_row;
            int global_col_B = tile_id_col + smem_col;

            float4* vec_tileB = (float4*)&tile_B[smem_row][smem_col];
            float4* vecB = (float4*)&matB[global_col_B + global_row_B * B.width];
            *vec_tileB = *vecB;
        }
        __syncthreads();
        for(int j=0;j<16;j++){
            float4 sub_a[4];
            float4 sub_b[4];
            for(int k=0;k<4;k++){
                sub_a[k] = *(float4*)&tile_A[tile_offset_row+k][j*4];
                sub_b[k] = *(float4*)&tile_B[j*4+k][tile_offset_col];
            }
            for(int m=0;m<4;m++){
                sum[m*4+0] += sub_a[m].x*sub_b[0].x;
                sum[m*4+0] += sub_a[m].y*sub_b[1].x;
                sum[m*4+0] += sub_a[m].z*sub_b[2].x;
                sum[m*4+0] += sub_a[m].w*sub_b[3].x;
                sum[m*4+1] += sub_a[m].x*sub_b[0].y;
                sum[m*4+1] += sub_a[m].y*sub_b[1].y;
                sum[m*4+1] += sub_a[m].z*sub_b[2].y;
                sum[m*4+1] += sub_a[m].w*sub_b[3].y;
                sum[m*4+2] += sub_a[m].x*sub_b[0].z;
                sum[m*4+2] += sub_a[m].y*sub_b[1].z;
                sum[m*4+2] += sub_a[m].z*sub_b[2].z;
                sum[m*4+2] += sub_a[m].w*sub_b[3].z;
                sum[m*4+3] += sub_a[m].x*sub_b[0].w;
                sum[m*4+3] += sub_a[m].y*sub_b[1].w;
                sum[m*4+3] += sub_a[m].z*sub_b[2].w;
                sum[m*4+3] += sub_a[m].w*sub_b[3].w;
            }
        }
        __syncthreads();
    }
    for(int i=0;i<4;i++){
        int row = tile_id_row + tile_offset_row + i;
        int col = tile_id_col + tile_offset_col;
        float4* vecC = (float4*)&matC[row * C.width + col];
        *vecC = *(float4*)&sum[i * 4];
    }
}

void useregister_tiled_matmul(int dim,Matrix A,Matrix B,Matrix C){
    Eval eval;
    float duration=eval.eval([&]() {
        dim3 grid(dim/64, dim/64);
        dim3 block(16,16);
        matmul3<<<grid,block>>>(A,B,C);
    });
    std::cout<<"Section:fragemnt_tiled_matmul================================================="<<std::endl;
    std::cout<<"duration: "<<duration<<" ms"<<std::endl;
    std::cout<<hipGetErrorString(hipGetLastError())<<std::endl;
    std::cout<<"isCalculationRight:"<<check(A,B,C)<<std::endl;
}

__global__ void matmul4(Matrix A, Matrix B, Matrix C) {
    __shared__ float tile_A[64][64];
    __shared__ float tile_B[64][64];


    float* matA = A.elements;
    float* matB = B.elements;
    float* matC = C.elements;

    int tile_id_row = blockIdx.y*64;
    int tile_id_col = blockIdx.x*64;
    int tile_offset_row = threadIdx.y*4;
    int tile_offset_col = threadIdx.x*4;
    float sum[16];

    #pragma unroll
    for(int i=0;i<16;i++){
        sum[i] = 0;
    }

    for(int i=0;i<gridDim.x;i++){
        for(int j=0;j<4;j++){
            int smem_row = tile_offset_row + j;
            int smem_col = tile_offset_col;

            int global_row_A = tile_id_row + smem_row;
            int global_col_A = i * 64 + smem_col;

            float4* vec_tileA = (float4*)&tile_A[smem_row][smem_col];
            float4* vecA = (float4*)&matA[global_col_A + global_row_A * A.width];
            *vec_tileA = *vecA;

            int global_row_B = i * 64 + smem_row;
            int global_col_B = tile_id_col + smem_col;

            float4* vec_tileB = (float4*)&tile_B[smem_row][smem_col];
            float4* vecB = (float4*)&matB[global_col_B + global_row_B * B.width];
            *vec_tileB = *vecB;
        }
        __syncthreads();
        for(int j=0;j<16;j++){
            float4 sub_a[4];
            float4 sub_b[4];
            for(int k=0;k<4;k++){
                sub_a[k] = *(float4*)&tile_A[tile_offset_row+k][j*4];
                {
                    sub_b[k] = *(float4*)&tile_B[j*4+k][tile_offset_col];
                }
            }
            for(int m=0;m<4;m++){
                sum[m*4+0] += sub_a[m].x*sub_b[0].x;
                sum[m*4+0] += sub_a[m].y*sub_b[1].x;
                sum[m*4+0] += sub_a[m].z*sub_b[2].x;
                sum[m*4+0] += sub_a[m].w*sub_b[3].x;
                sum[m*4+1] += sub_a[m].x*sub_b[0].y;
                sum[m*4+1] += sub_a[m].y*sub_b[1].y;
                sum[m*4+1] += sub_a[m].z*sub_b[2].y;
                sum[m*4+1] += sub_a[m].w*sub_b[3].y;
                sum[m*4+2] += sub_a[m].x*sub_b[0].z;
                sum[m*4+2] += sub_a[m].y*sub_b[1].z;
                sum[m*4+2] += sub_a[m].z*sub_b[2].z;
                sum[m*4+2] += sub_a[m].w*sub_b[3].z;
                sum[m*4+3] += sub_a[m].x*sub_b[0].w;
                sum[m*4+3] += sub_a[m].y*sub_b[1].w;
                sum[m*4+3] += sub_a[m].z*sub_b[2].w;
                sum[m*4+3] += sub_a[m].w*sub_b[3].w;
            }
        }
        __syncthreads();
    }
    for(int i=0;i<4;i++){
        int row = tile_id_row + tile_offset_row + i;
        int col = tile_id_col + tile_offset_col;
        float4* vecC = (float4*)&matC[row * C.width + col];
        *vecC = *(float4*)&sum[i * 4];
    }
}

void bc_free_tiled_matmul(int dim,Matrix A,Matrix B,Matrix C){
    Eval eval;
    float duration=eval.eval([&]() {
        dim3 grid(dim/64, dim/64);
        dim3 block(16,16);
        matmul4<<<grid,block>>>(A,B,C);
    });
    std::cout<<"Section:fragemnt_tiled_matmul================================================="<<std::endl;
    std::cout<<"duration: "<<duration<<" ms"<<std::endl;
    std::cout<<hipGetErrorString(hipGetLastError())<<std::endl;
    std::cout<<"isCalculationRight:"<<check(A,B,C)<<std::endl;
}

int main(int argc,char* argv[]) {
    int dim =1024;
    if(argc>1) {
        dim = atoi(argv[1]);
    }
    assert(dim%128==0);

    Matrix A,B,C;
    initializeMatrix(&A, dim);
    initializeMatrix(&B, dim);
    initializeMatrix(&C, dim);
    basic_tiled_matmul(dim,A,B,C);
    fragent_vec_tiled_matmul(dim,A,B,C);
    useregister_tiled_matmul(dim,A,B,C);
    bc_free_tiled_matmul(dim,A,B,C);
}