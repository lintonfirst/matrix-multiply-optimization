#include <hip/hip_runtime.h>
#include <assert.h>
#include <cstdlib>
#include "common/matrix.h"
#include "common/eval.h"
#include "common/check.h"
#include <iostream>

int main(int argc,char* argv[]) {
    int dim =1024;
    if(argc>1) {
        dim = atoi(argv[1]);
    }
    assert(dim%128==0);

    Matrix A,B,C;
    initializeMatrix(&A, dim);
    initializeMatrix(&B, dim);
    initializeMatrix(&C, dim);
    Eval eval;

    float duration=eval.eval([&]() {
        
    });
    std::cout<<"duration: "<<duration<<" ms"<<std::endl;
}